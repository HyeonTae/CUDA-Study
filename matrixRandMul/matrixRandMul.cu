#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <time.h>

#define N 100
__global__ void mul(int a[][N], int b[][N], int c[][N]){
	int row = blockIdx.x;
	int col = blockIdx.y;

	if(row < N && col < N) {
		for(int i = 0; i < N; i++) {
			for(int j = 0; j < N; j++) {
				c[i][j] = 0;
				for(int k = 0; k < N; k++) {
					c[i][j] += a[i][k] * b[k][j];
				}
			}
		}
	}
}

int main(){
	int (*pa)[N], (*pb)[N], (*pc)[N];
	int a[N][N], b[N][N], c[N][N];

	srand((unsigned)time(NULL));	

	hipEvent_t start, stop;

	hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
	
	hipMalloc((void**)&pa, (N*N) * sizeof(int));
	hipMalloc((void**)&pb, (N*N) * sizeof(int));
	hipMalloc((void**)&pc, (N*N) * sizeof(int));

	for(int i = 0 ; i<N ; i++){
		for(int j = 0 ; j<N ; j++) {
			a[i][j] = rand()%10 + 1;
			b[i][j] = rand()%10 + 1;
		}
	}
	
	hipMemcpy(pa, a, (N*N) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pb, b, (N*N) * sizeof(int), hipMemcpyHostToDevice);	
	
	dim3 blocksPerBlock(N,N);
	mul<<<blocksPerBlock,1>>>(pa, pb, pc);

	hipMemcpy(c, pc, (N*N) * sizeof(int), hipMemcpyDeviceToHost);

	printf("matrix multiplication per block\n");

/*	for(int i = 0 ; i<N ; i++){
		for(int j = 0 ; j<N ; j++) {
			printf("%d ",c[i][j]);
		}
		printf("\n");
	}
*/
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Time to generate : %3.1f ms\n", elapsedTime);

	hipFree(pa);
	hipFree(pb);
	hipFree(pc);

	return 0;

}
